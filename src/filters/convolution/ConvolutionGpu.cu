#include "hip/hip_runtime.h"
#include "filters/convolution/ConvolutionGpu.cuh"
#include <hip/hip_runtime.h>
#include <>
#include <iostream>
#include <stdexcept>

__device__ uint8_t get_channel(const uint8_t* input, int x, int y, int channel, int width, int height, int pad, PaddingType paddingtype) {
    if (x < 0 || y < 0 || x >= width || y >= height) {
        if (paddingtype == PaddingType::ZERO)
            return 0;
        if (paddingtype == PaddingType::REPLICATE) {
            x = max(0, min(x, width - 1));
            y = max(0, min(y, height - 1));
        }
    }
    return input[(y * width + x) * 4 + channel];
}

__global__ void convolutionKernel(uint8_t* input, uint8_t* output, double* kernel, int width, int height, int kernelSize, PaddingType paddingtype) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width || y >= height) return;

    int pad = kernelSize / 2;
    double sum_r = 0.0, sum_g = 0.0, sum_b = 0.0;

    for (int ky = -pad; ky <= pad; ++ky) {
        for (int kx = -pad; kx <= pad; ++kx) {
            double k_val = kernel[(ky + pad) * kernelSize + (kx + pad)];
            sum_r += get_channel(input, x + kx, y + ky, 0, width, height, pad, paddingtype) * k_val;
            sum_g += get_channel(input, x + kx, y + ky, 1, width, height, pad, paddingtype) * k_val;
            sum_b += get_channel(input, x + kx, y + ky, 2, width, height, pad, paddingtype) * k_val;
        }
    }

    int idx = (y * width + x) * 4;
    output[idx]     = min(max(int(sum_r), 0), 255);
    output[idx + 1] = min(max(int(sum_g), 0), 255);
    output[idx + 2] = min(max(int(sum_b), 0), 255);
    output[idx + 3] = 255; // Alpha inchangé
}

void computeGPUNaiveConvolution(std::vector<std::uint8_t>& pixels, sf::Vector2u size, const std::vector<double>& kernel, size_t kernel_size, PaddingType paddingtype) {
    if (kernel_size % 2 == 0 || kernel.size() != kernel_size * kernel_size) {
        throw std::invalid_argument("The kernel must be odd and squared.");
    }

    int imgSize = size.x * size.y * 4;
    uint8_t* d_input = nullptr;
    uint8_t* d_output = nullptr;
    double* d_kernel = nullptr;

    hipMalloc(&d_input, imgSize);
    hipMalloc(&d_output, imgSize);
    hipMalloc(&d_kernel, kernel.size() * sizeof(double));

    hipMemcpy(d_input, pixels.data(), imgSize, hipMemcpyHostToDevice);
    hipMemcpy(d_kernel, kernel.data(), kernel.size() * sizeof(double), hipMemcpyHostToDevice);

    dim3 threads(16, 16);
    dim3 blocks((size.x + threads.x - 1) / threads.x, (size.y + threads.y - 1) / threads.y);
    convolutionKernel<<<blocks, threads>>>(d_input, d_output, d_kernel, size.x, size.y, kernel_size, paddingtype);
    hipDeviceSynchronize();

    hipMemcpy(pixels.data(), d_output, imgSize, hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);
    hipFree(d_kernel);
}
